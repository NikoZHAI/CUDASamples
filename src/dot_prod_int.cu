#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

const size_t nThreadsPerBlock = 128;


static void HandleError(hipError_t err, const char *file, int line )
{
    if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
            file, line );
    exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// Direct use of global memory between threads may be very wrong?
__global__ void dotProd(int length, int *u, int *v, int *out) {
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned tid_const = threadIdx.x + blockDim.x * blockIdx.x;
    int temp = 0;

    while (tid < length) {
        temp += u[tid] * v[tid];
        tid += blockDim.x * gridDim.x;
    }
    out[tid_const] = temp;
}


__global__ void dotProdWithSharedMem(int length, int *u, int *v, int *out) {
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned cid = threadIdx.x;
    __shared__ int cache[nThreadsPerBlock];
    int temp = 0;

    while (tid < length) {
        temp += u[tid] * v[tid];
        tid  += blockDim.x * gridDim.x;
    }

    cache[cid] = temp;
    __syncthreads();
    
    int i = blockDim.x/2;
    while (i != 0) {
        if (cid < i) {
            cache[cid] += cache[cid + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cid == 0) {
        out[blockIdx.x] = cache[0];
    }
}


int main(int argc, char* argv[]) {
    
    size_t vec_len  = 1 << std::stoi(argv[1]);
    size_t size     = vec_len * sizeof(int);
    size_t nthreads = std::stoi(argv[2]);
    size_t nblocks  = std::stoi(argv[3]);
    // size_t nblocks  = (vec_len + nthreads - 1) / nthreads;
    size_t size_out   = nthreads*nblocks*sizeof(int);
    size_t size_out_2 = nblocks*sizeof(int);

    int *u     = (int *)malloc(size);
    int *v     = (int *)malloc(size);
    int *out   = (int *)malloc(size_out);
    int *out_2 = (int *)malloc(size_out_2);
    
    int *dev_u, *dev_v, *dev_out, *dev_out_2; // Device arrays
   
    int res_gpu = 0;
    int res_gpu_2 = 0;
    int res_cpu = 0;

    dim3 dimGrid(nblocks, 1, 1);
    dim3 dimBlocks(nthreads, 1, 1);

    // Initiate values
    for(size_t i=0; i<vec_len; ++i) {
        u[i] = i;
        v[i] = i;
    }

    HANDLE_ERROR( hipMalloc((void**)&dev_u, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_v, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out, size_out) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out_2, size_out_2) );
    HANDLE_ERROR( hipMemcpy(dev_u, u, size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_v, v, size, hipMemcpyHostToDevice) );
    //HANDLE_ERROR( cudaMemset(dev_out, 0, size_out) );
    //HANDLE_ERROR( cudaMemset(dev_out_2, 0, size_out_2) );



    auto t1_gpu = std::chrono::system_clock::now();

    dotProd <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out);

    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out, dev_out, size_out, hipMemcpyDeviceToHost) );

    // Reduction
    for(size_t i=0; i<nthreads*nblocks; ++i) {
        res_gpu += out[i];
    }

    auto t2_gpu = std::chrono::system_clock::now();

    
    // CPU version for result-check
    for(size_t i=0; i<vec_len; ++i) {
        res_cpu += u[i] * v[i];
    }
    auto t2_cpu = std::chrono::system_clock::now();

    // GPU version with shared memory
    dotProdWithSharedMem <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out_2);

    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out_2, dev_out_2, size_out_2, hipMemcpyDeviceToHost) );

    // Reduction
    for(size_t i=0; i<nblocks; ++i) {
        res_gpu_2 += out_2[i];
    }
    auto t2_gpu_2 = std::chrono::system_clock::now();

    double t_gpu   = std::chrono::duration <double, std::milli> (t2_gpu - t1_gpu).count();
    double t_gpu_2 = std::chrono::duration <double, std::milli> (t2_gpu_2 - t2_cpu).count();
    double t_cpu   = std::chrono::duration <double, std::milli> (t2_cpu - t2_gpu).count();

    printf("GPU result: %i, time consummed: %.5f ms\n", res_gpu, t_gpu);
    printf("GPU result: %i, time consummed: %.5f ms\n", res_gpu_2, t_gpu_2);
    printf("CPU result: %i, time consummed: %.5f ms\n", res_cpu, t_cpu);

    hipFree(dev_u);
    hipFree(dev_v);
    hipFree(dev_out);
    hipFree(dev_out_2);
    free(u);
    free(v);
    free(out);
    free(out_2);

    return 0;
}

