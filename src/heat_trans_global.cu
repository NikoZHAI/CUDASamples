#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <common/book.h>
#include <common/cpu_anim.h>


#ifndef NTHREADS
#define NTHREADS (int) 16
#endif

#ifndef DIM
#define DIM (int) 256
#endif

#ifndef NSTEPS
#define NSTEPS (int) 90
#endif

#ifndef SPEED
#define SPEED 0.25f
#endif

#ifndef MAX_TEMP
#define MAX_TEMP 1.f
#endif

#ifndef MIN_TEMP
#define MIN_TEMP 0.0001f
#endif


// Declarations
struct DataBlock;
__global__ void copy_const_kernel( float*, const float* );
__global__ void step_run_kernel( float*, const float* );
__global__ void my_float_to_color( unsigned char*, const float* ); // float to color defined in book.h, line 80
void my_swap( float**, float** );
void anim_exit_callback( DataBlock* );
void anim_gpu( DataBlock*, int );

struct DataBlock {
    CPUAnimBitmap   *bitmap;
    unsigned char   *dev_bitmap;
    float           *dev_constSrc;
    float           *dev_inSrc;
    float           *dev_outSrc;

    hipEvent_t     start;
    hipEvent_t     stop;
    float           totalElapsedTime;
    float           frames;
};


int main (void) {

    DataBlock       data;
    CPUAnimBitmap   bitmap( DIM, DIM, &data );
    
    data.bitmap = &bitmap;
    data.frames = 0.f;
    data.totalElapsedTime = 0.f;

    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );
    // Note that one char takes 1 byte, one float takes 4 bytes, 
    // so the following allocation is correct
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_bitmap)  , bitmap.image_size() ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_constSrc), bitmap.image_size() ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_inSrc)   , bitmap.image_size() ) );
    HANDLE_ERROR( hipMalloc( (void**)&(data.dev_outSrc)  , bitmap.image_size() ) );

    /* Initial Condition */
    float *cond_init = (float*) malloc( bitmap.image_size() );
    for (int i=0; i<DIM*DIM; i++) {
        cond_init[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x>300) && (x<600) && (y>310) && (y<601))
            cond_init[i] = MAX_TEMP;
    }
    cond_init[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
    cond_init[DIM*700+100] = MIN_TEMP;
    cond_init[DIM*300+300] = MIN_TEMP;
    cond_init[DIM*200+700] = MIN_TEMP;
    for (int y=800; y<900; y++) {
        for (int x=400; x<500; x++) {
            cond_init[x+y*DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy(data.dev_constSrc,
                             cond_init,
                             bitmap.image_size(), 
                             hipMemcpyHostToDevice) );

    for (int y=800; y<DIM; ++y) {
        for (int x=0; x<200; ++x) {
            cond_init[x+y*DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy(data.dev_inSrc,
                             cond_init,
                             bitmap.image_size(), 
                             hipMemcpyHostToDevice) );
    free( cond_init );

    bitmap.anim_and_exit( (void (*)(void*, int))anim_gpu,
                          (void (*)(void*))anim_exit_callback );

}


void anim_gpu ( DataBlock *d, int ticks ) {

    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );

    dim3    dimGrid( DIM/NTHREADS, DIM/NTHREADS );
    dim3    dimBlocks( NTHREADS, NTHREADS );

    for (int i=0; i<NSTEPS; ++i) {
        copy_const_kernel <<< dimGrid, dimBlocks >>> ( d->dev_inSrc,
                                                       d->dev_constSrc );
        step_run_kernel <<< dimGrid, dimBlocks >>> ( d->dev_outSrc, d->dev_inSrc );
        my_swap( &d->dev_inSrc, &d->dev_outSrc );
    }
    my_float_to_color <<< dimGrid, dimBlocks >>> ( d->dev_bitmap, d->dev_inSrc );
    hipDeviceSynchronize();

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );

    float elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, d->start, d->stop ) );
    
    d->totalElapsedTime += elapsedTime;
    ++(d->frames);
    
    printf( "Mean computation time per frame: %3.2f ms\n",
            d->totalElapsedTime/d->frames );
}


void anim_exit_callback ( DataBlock* d ) {
    hipFree( d->dev_bitmap );
    hipFree( d->dev_inSrc  );
    hipFree( d->dev_outSrc );
    hipFree( d->dev_constSrc );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


__global__ void copy_const_kernel( float *inSrc, const float *constSrc ) {
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int offset = x + y * blockDim.x * gridDim.x;

    if ( constSrc[offset] != 0 ) { inSrc[offset] = constSrc[offset]; }
}


__global__ void step_run_kernel( float *outSrc, const float *inSrc ) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    /* AND THAT, IS WHAT PPL CALLED STUPID */
    // int x1  = (x!=(DIM-1)) ? (x+1) : x; // Right
    // int x4  = x;                        // Bottom
    // int x16 = x ? (x-1) : x;            // Left
    // int x64 = x;                        // Top

    // int y1  = y;
    // int y4  = y ? (y-1) : y;
    // int y16 = y;
    // int y64 = (y!=(DIM-1)) ? (y+1) : y;

    // int offset1  = x1 + y1 * blockDim.x * gridDim.x;
    // int offset4  = x4 + y4 * blockDim.x * gridDim.x;
    // int offset16 = x16 + y16 * blockDim.x * gridDim.x;
    // int offset64 = x64 + y64 * blockDim.x * gridDim.x;

    // outSrc[offset] = ( 1.f - 4.f * SPEED ) * inSrc[offset] +
    //                  SPEED * ( inSrc[offset1] + inSrc[offset4] +
    //                            inSrc[offset16] + inSrc[offset64] );
    /* END OF STUPIDITY */

    int top    =      y       ? (offset-DIM) :  offset;
    int right  = (x != DIM-1) ? (offset + 1) :  offset;
    int bottom = (y != DIM-1) ? (offset+DIM) :  offset;
    int left   =      x       ? (offset - 1) :  offset;

    outSrc[offset] = ( 1.f - 4.f * SPEED ) * inSrc[offset] + 
                     SPEED * ( inSrc[top] + inSrc[right] + inSrc[bottom] + inSrc[left]);
}


__global__ void my_float_to_color(unsigned char *ptr, const float *inSrc) {
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int offset = x + y * blockDim.x * gridDim.x;

    ptr[offset*4 + 0] = (int)( 255 * inSrc[offset] );
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 0;
}


void my_swap (float **in, float **out) {
    float *dummy = *out;
    *out = *in;
    *in = dummy;
}
