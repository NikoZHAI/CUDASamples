#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>


#define PI 3.1415926535897932f

const size_t nThreadsPerBlock = 256;


static void HandleError(hipError_t err, const char *file, int line )
{
    if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
            file, line );
    exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void dotProd(int length, float *u, float *v, float *out) {
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned tid_const = threadIdx.x + blockDim.x * blockIdx.x;
    float temp = 0;

    while (tid < length) {
        temp += u[tid] * v[tid];
        tid  += blockDim.x * gridDim.x;
    }
    out[tid_const] = temp;
}


__global__ void dotProdWithSharedMem(int length, float *u, float *v, float *out) {
    __shared__ float cache[nThreadsPerBlock];
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned cid = threadIdx.x;

    float temp = 0;
    while (tid < length) {
        temp += u[tid] * v[tid];
        tid  += blockDim.x * gridDim.x;
    }

    cache[cid] = temp;
    __syncthreads();
    
    int i = nThreadsPerBlock/2;
    while (i != 0) {
        if (cid < i) {
            cache[cid] += cache[cid + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cid == 0) {
        out[blockIdx.x] = cache[0];
    }
}


int main(int argc, char* argv[]) {
    
    size_t vec_len  = 1 << std::stoi(argv[1]);
    size_t size     = vec_len * sizeof(float);
    size_t nblocks  = std::stoi(argv[2]);
    size_t size_out   = nThreadsPerBlock*nblocks*sizeof(float);
    size_t size_out_2 = nblocks*sizeof(float);

    float *u     = (float *)malloc(size);
    float *v     = (float *)malloc(size);
    float *out   = (float *)malloc(size_out);
    float *out_2 = (float *)malloc(size_out_2);
    
    float *dev_u, *dev_v, *dev_out, *dev_out_2; // Device arrays
   
    float res_gpu = 0;
    float res_gpu_2 = 0;
    float res_cpu = 0;

    dim3 dimGrid(nblocks, 1, 1);
    dim3 dimBlocks(nThreadsPerBlock, 1, 1);

    // Initiate values
    for(size_t i=0; i<vec_len; ++i) {
        u[i] = std::sin(i*PI*1E-2);
        v[i] = std::cos(i*PI*1E-2);
    }

    HANDLE_ERROR( hipMalloc((void**)&dev_u, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_v, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out, size_out) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out_2, size_out_2) );
    HANDLE_ERROR( hipMemcpy(dev_u, u, size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_v, v, size, hipMemcpyHostToDevice) );


    auto t1_gpu = std::chrono::system_clock::now();
    dotProd <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out);
    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out, dev_out, size_out, hipMemcpyDeviceToHost) );
    // Reduction
    for(size_t i=0; i<nThreadsPerBlock*nblocks; ++i) {
        res_gpu += out[i];
    }


    auto t2_gpu = std::chrono::system_clock::now();
    // GPU version with shared memory
    dotProdWithSharedMem <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out_2);
    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out_2, dev_out_2, size_out_2, hipMemcpyDeviceToHost) );
    // Reduction
    for(size_t i=0; i<nblocks; ++i) {
        res_gpu_2 += out_2[i];
    }
    auto t3_gpu = std::chrono::system_clock::now();


    // CPU version for result-check
    for(size_t i=0; i<vec_len; ++i) {
        res_cpu += u[i] * v[i];
    }
    auto t2_cpu = std::chrono::system_clock::now();


    double t_gpu = std::chrono::duration <double, std::milli> (t2_gpu - t1_gpu).count();
    double t_gpu_2 = std::chrono::duration <double, std::milli> (t3_gpu - t2_gpu).count();
    double t_cpu = std::chrono::duration <double, std::milli> (t2_cpu - t3_gpu).count();

    printf("Number of threads per block : %i \n", nThreadsPerBlock);
    printf("Number of blocks in the grid: %i \n", nblocks);
    printf("Total number of threads     : %i \n", nThreadsPerBlock*nblocks);
    printf("Length of vectors           : %i \n\n", vec_len);
    printf("GPU using registers: %.10f, time consummed: %.5f ms\n", res_gpu, t_gpu);
    printf("GPU using shared   : %.10f, time consummed: %.5f ms\n", res_gpu_2, t_gpu_2);
    printf("CPU result         : %.10f, time consummed: %.5f ms\n", res_cpu, t_cpu);

    hipFree(dev_u);
    hipFree(dev_v);
    hipFree(dev_out);
    hipFree(dev_out_2);
    free(u);
    free(v);
    free(out);
    free(out_2);

    return 0;
}

