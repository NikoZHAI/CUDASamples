#include <iostream>
#include <string>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>


#define PI 3.1415927

const size_t nThreadsPerBlock = 256;


static void HandleError(hipError_t err, const char *file, int line )
{
    if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
    file, line );
    exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// Direct use of global memory between threads may be very wrong?
__global__ void dotProd(int length, float *u, float *v, float *out) {
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned tid_const = threadIdx.x + blockDim.x * blockIdx.x;
    float temp = 0.f;

    while (tid < length) {
        temp += u[tid] * v[tid];
        tid += blockDim.x * gridDim.x;
    }
    __syncthreads();
    out[tid_const] = temp;
}


__global__ void dotProdWithSharedMem(int length, float *u, float *v, float *out) {
    unsigned tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned cid = threadIdx.x;
    __shared__ float cache[nThreadsPerBlock];
    float temp = 0.f;

    while (tid < length) {
        temp += u[tid] * v[tid];
        tid  += blockDim.x * gridDim.x;
    }

    cache[cid] = temp;
    __syncthreads();
    
    int i = blockDim.x/2;
    while (i != 0) {
        if (cid < i) {
            cache[cid] += cache[cid + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cid == 0) {
        out[blockIdx.x] = cache[0];
    }
}


int main(int argc, char* argv[]) {
    
    size_t vec_len  = 1 << std::stoi(argv[1]);
    size_t size     = vec_len * sizeof(float);
    size_t nthreads = std::stoi(argv[2]);
    size_t nblocks  = std::stoi(argv[3]);
    // size_t nblocks  = (vec_len + nthreads - 1) / nthreads;
    size_t size_out   = nthreads*nblocks*sizeof(float);
    size_t size_out_2 = nblocks*sizeof(float);

    float *u   = (float *)malloc(size);
    float *v   = (float *)malloc(size);
    float *out = (float *)malloc(size_out);
    float *out_2 = (float *)malloc(size_out_2);
    
    float *dev_u, *dev_v, *dev_out, *dev_out_2; // Device arrays
   
    float res_gpu = 0;
    float res_gpu_2 = 0;
    float res_cpu = 0;

    dim3 dimGrid(nblocks, 1, 1);
    dim3 dimBlocks(nthreads, 1, 1);

    // Initiate values
    for(size_t i=0; i<vec_len; ++i) {
        u[i] = std::sin(i*PI*1E-2);
        v[i] = std::cos(i*PI*1E-2);
    }

    HANDLE_ERROR( hipMalloc((void**)&dev_u, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_v, size) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out, size_out) );
    HANDLE_ERROR( hipMalloc((void**)&dev_out_2, size_out_2) );
    HANDLE_ERROR( hipMemcpy(dev_u, u, size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_v, v, size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemset(dev_out, 0.f, size_out) );
    HANDLE_ERROR( hipMemset(dev_out_2, 0.f, size_out_2) );



    auto t1_gpu = std::chrono::system_clock::now();

    dotProd <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out);

    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out, dev_out, size_out, hipMemcpyDeviceToHost) );

    // Reduction
    for(size_t i=0; i<nthreads*nblocks; ++i) {
        res_gpu += out[i];
    }

    auto t2_gpu = std::chrono::system_clock::now();

    
    // CPU version for result-check
    for(size_t i=0; i<vec_len; ++i) {
        res_cpu += u[i] * v[i];
    }
    auto t2_cpu = std::chrono::system_clock::now();

    // GPU version with shared memory
    dotProdWithSharedMem <<<dimGrid, dimBlocks>>> (vec_len, dev_u, dev_v, dev_out_2);

    hipDeviceSynchronize();
    HANDLE_ERROR( hipMemcpy(out_2, dev_out_2, size_out_2, hipMemcpyDeviceToHost) );

    // Reduction
    for(size_t i=0; i<nblocks; ++i) {
        res_gpu_2 += out_2[i];
    }
    auto t2_gpu_2 = std::chrono::system_clock::now();

    double t_gpu   = std::chrono::duration <double, std::milli> (t2_gpu - t1_gpu).count();
    double t_gpu_2 = std::chrono::duration <double, std::milli> (t2_gpu_2 - t2_cpu).count();
    double t_cpu   = std::chrono::duration <double, std::milli> (t2_cpu - t2_gpu).count();

    printf("GPU result: %.10f, time consummed: %.5f ms\n", res_gpu, t_gpu);
    printf("GPU result: %.10f, time consummed: %.5f ms\n", res_gpu_2, t_gpu_2);
    printf("CPU result: %.10f, time consummed: %.5f ms\n", res_cpu, t_cpu);

    hipFree(dev_u);
    hipFree(dev_v);
    hipFree(dev_out);
    free(u);
    free(v);
    free(out);

    return 0;
}

