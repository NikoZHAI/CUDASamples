#include <stdio.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <chrono>

int tid;
double pi = 0;

static void HandleError( hipError_t err,
						 const char *file,
						 int line )
{
	if (err != hipSuccess) {
	printf( "%s in %s at line %d\n", hipGetErrorString( err ),
	file, line );
	exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, int nthreads, int nblocks) {
	int i;
	double x;

	// Sequential thread index across the blocks
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	
	for (i=idx; i < nbin; i+=nthreads*nblocks) {
		x = (i+.5) / nbin;
		sum[idx] += 4./(1. + x*x);
	}
}

// Main routine that executes on the host
int main(int argc, char* argv[]) {

	/* Settings */
	unsigned int n_steps  = 1<<std::stoi(argv[1]);
	unsigned int nblocks  = std::stoi(argv[2]);
    unsigned int nthreads = std::stoi(argv[3]);
    unsigned int nStreams = std::stoi(argv[4]);

	printf("         N          = %11i\n", n_steps);
	printf("  N thread blocks   = %11i\n", nblocks);
	printf("N threads per block = %11i\n", nblocks);

	dim3 dimGrid(nblocks,1,1);  	// Grid dimensions
	dim3 dimBlock(nthreads,1,1);    // Block dimensions
	double *sumHost, *sumDev;  		// Pointer to host & device arrays

	size_t size = nblocks*nthreads*sizeof(double);  // Size of the device array

	sumHost = (double *)malloc(size);  //  Allocate array on host
	HANDLE_ERROR(hipMalloc((void **) &sumDev, size));  // Allocate array on device
	
	auto t1 = std::chrono::system_clock::now();
	// Initialization
	HANDLE_ERROR(hipMemset(sumDev, 0., size));
	
	/* Invoke the CUDA kernel */
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, n_steps, nthreads, nblocks); // call CUDA kernel
	hipDeviceSynchronize();  // Wait for calculations finished

	/* Reduction */
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<nblocks*nthreads; ++tid) {
		pi += sumHost[tid];
	}
	pi /= n_steps;
	auto t2 = std::chrono::system_clock::now();

	/* Results */
	printf("\nPI = %1.18f\n",pi);
	std::cout << "Computation time: "
			  << std::chrono::duration <double, std::milli> (t2 - t1).count()
			  << " ms.\n";

	// Free memory
	free(sumHost); 
	hipFree(sumDev);

	return 0;
}

